#include "hip/hip_runtime.h"
#include <iostream>
#include <math.h>
#include <algorithm>
#include <cassert>
#include <string>
#include <fstream>
#include <vector>

using namespace std;
using std::min;
using std::max;



bool notAllVerticesColored(int* C, int n, int &count)
{
    bool result = false;
    count = n;
    for(int i = 1; i <= n; i++)
    {
        if(C[i] == 0)
        {
            // return true;
            result = true;
            count--;
        }
    }
    return result;
}

bool checkValidColoring(int* graph, int* adjList, int* C, int n)
{
    cout << "coloring" << endl;
    for(int i=1;i<=n;i++)
    {
        cout << i << " " << C[i] << endl;
        for(int j=graph[i];j<graph[i+1];j++)
        {
            if(C[adjList[j]] == C[i])
            {
                return false;
            }
        }
    }
    return true;
}

void parseInput(char* inputFile, int &n, int &m, int* &graph, int* &adjList, int &D)
{
    // TODO aditya: malloc the graph here after finding the number of edges and length of array needed
    // Determine the number of colors D needed
    fstream f;
    f.open(inputFile, ios::in);
    f >> n >> m;

    graph = new int[n + 2];
    adjList = new int[2*m];
    graph[0] = m;
    vector<int> g[n + 1];
    int u,v;
    for(int i = 0; i < m; i ++)
    {
        f >> u >> v;
        g[u].push_back(v);
        g[v].push_back(u);
    }
    D = 0;
    int ctr = 0;
    for(int i = 1; i <=n; i ++)
    {
        graph[i] = ctr;
        for(auto &x: g[i])
        {
            adjList[ctr++] = x;
        }
        D = max(D, (int)g[i].size());
    }
    D++;
    graph[n + 1] = ctr;
    // for(int i = 0; i <2*m ; i++)
    // {
    //     cout << adjList[i] <<" ";
    // }
    // cout << endl;
    // for(int i = 0; i < n+2; i ++)
    // {
    //     cout << graph[i] <<" ";
    // }
    // cout << endl;

}
int* getRho(int* graph, int* adjList, int strategy, int n)
{
    // this has to give the total order permutation on the vertices.
    // based on strategy should give random order or adg order or dec-adg order
    // Should we implement this in Device using parallel programming
    // Maybe but later

    int *rho = new int[n + 1];
    rho[0] = -1;
    for(int i  = 1; i <= n; i ++)
    {
        rho[i] = i;
    }
    random_shuffle(rho + 1, rho + n + 1);
    return rho;
}

__device__ int getColor(int* graph, int* adjList, int* rho, int* C, int v, int D)
{
    bool* B = new bool[D+1]();
    memset(B,0,sizeof(bool)*(D+1));
    // if(v==n) TODO make sure n+1 th entry should be the end index of the array to make sure this works.
    // This is very important as we do not want if statement here as this function gets used a lot of times
    for(int i = graph[v]; i < graph[v+1]; i++)
    {
        if(rho[adjList[i]] > rho[v])
        {
            if(C[adjList[i]] == 0)
            {
                free(B);
                return 0;
            }
            else
            {
                B[C[adjList[i]]] = true;
            }
        }
    }
    // printf("get color %d\n",v);
    for(int i =1 ;i <= D; i++)
    {
        if(!B[i])
        {
            free(B);
            return i;
        }
    }
    // Should not come here at all
    assert(false);
    free(B);
    return 0;
}

__global__ void jpadg(int* graph, int* adjList, int* rho, int* C, int D, int n)
{
    int u = (blockDim.x * blockIdx.x)+threadIdx.x+1;
    if(u> n || u<1)
    {
        return;
    }
    int minC = 0;
    // A very important change to make while loop to if. We just give one chance if doesn't get color
    // give another chance again later, no point in giving a chance again immediately
    if(C[u] == 0)
    {
        minC = getColor(graph, adjList, rho, C, u, D);
        if(minC != 0)
        {
            C[u] = minC;
        }
    }
    return;
}


int main(int argc, char** argv)
{
    if(argc == 1)
    {
        cout << "No input" << endl;
        return 0;
    }
    // TODO by aditya
    int n, m, D;
    int *adjList = NULL; //This is the adjacency list
    int *graph = NULL;
    cout << "Parse inp" << endl;
    parseInput(argv[1], n , m, graph, adjList, D);
    cout << "Parse input D" << D << " n " << n << " m " << m << endl;
    int* rho = getRho(graph, adjList, 1, n); // 1= random order or largest degree first
    cout << "Get Rho" << endl;
    dim3 gridDim((n+1023)/1024,1,1);
    dim3 blockDim(1024,1,1);

    int* C = new int[n+1]();
    memset(C, 0, sizeof(int)*(n+1));

    int *d_graph, *d_adjList, *d_rho, *d_C;

    if(hipMalloc(&d_graph,sizeof(int)*(n+2))!=hipSuccess)
    {
        cout << "Could not allocate d_graph" << endl;
    }

    if(hipMalloc(&d_adjList, sizeof(int)*(2*m))!=hipSuccess)
    {
        cout << "Could not allocate d_adjList" << endl;
    }

    if(hipMalloc(&d_rho,sizeof(int)*(n+1))!=hipSuccess)
    {
        cout << "Could not allocate d_graph" << endl;
    }

    if(hipMalloc(&d_C, sizeof(int)*(n+1))!=hipSuccess)
    {
        cout << "Could not allocate d_C" << endl;
    }

    if(hipMemcpy(d_graph,graph,sizeof(int)*(n+2),hipMemcpyHostToDevice) != hipSuccess){
        cout<<"Could not copy graph into d_graph"<<endl;
    }

    if(hipMemcpy(d_adjList,adjList,sizeof(int)*(2*m),hipMemcpyHostToDevice) != hipSuccess){
        cout<<"Could not copy adjList into d_adjList"<<endl;
    }

    if(hipMemcpy(d_rho,rho,sizeof(int)*(n+1),hipMemcpyHostToDevice) != hipSuccess){
        cout<<"Could not copy rho into d_rho"<<endl;
    }

    if(hipMemset(d_C, 0, sizeof(int)*(n+1)) != hipSuccess)
    {
        cout << "Could not memset C" << endl;
    }

    int iter = 0;
    int count = 0;

    while(notAllVerticesColored(C,n,count))
    {
        // We need not run again for all vertices
        // Run only for uncolored vertices VERY IMPORTANT
        cout << "Running iteration " << iter++ << " colored : " << count << "/" << n << endl;
        jpadg<<<gridDim, blockDim>>>(d_graph, d_adjList, d_rho, d_C, D, n);
        auto code = hipMemcpy(C,d_C,sizeof(int)*(n+1),hipMemcpyDeviceToHost);
        if (code != hipSuccess)
        {
            cout << "GPUassert:" << hipGetErrorName(code) << " " <<  hipGetErrorString(code) << " " << endl;
        }
        // for(int i = 1;i<=n;i++)
        // {
        //     cout << "color of " << i << " " << C[i] << endl;
        // }
    }

    hipFree(d_graph);
    hipFree(d_adjList);
    hipFree(d_rho);
    hipFree(d_C);

    free(rho);
    assert(checkValidColoring(graph, adjList, C, n));
    free(graph);
    free(adjList);
    free(C);
    return 0;
}