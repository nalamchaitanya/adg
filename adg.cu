#include "hip/hip_runtime.h"
#include <iostream>
#include <math.h>
#include<hiprand.h>
#include<hiprand/hiprand_kernel.h>
#include <algorithm>
#include <cassert>
#include <string>
#include <fstream>
#include <vector>
#include<cstdio>
#include<map>

using namespace std;
using std::min;
using std::max;
const long scale_1 = 1e16,scale_2 = 1e15;

// bool notAllVerticesOrdered(long* ordering, int n)
// {
//     bool result = false;
//     for(int i = 1; i <= n; i++)
//     {
//         if(ordering[i] == 0)
//         {
//             // return true;
//             // TODO return here, are we using count somewhere?
//             return true;
//         }
//     }
//     return result;
// }

// bool notAllVerticesColored(int* C, int n, int &count)
// {
//     bool result = false;
//     count = n;
//     for(int i = 1; i <= n; i++)
//     {
//         if(C[i] == 0)
//         {
//             // return true;
//             result = true;
//             count--;
//         }
//     }
//     return result;
// }

int checkValidColoring(int* graph, int* adjList, int* C, int n)
{
    // cout << "coloring" << endl;
    int maxcolor = 0;
    for(int i=1;i<=n;i++)
    {
        maxcolor = max(maxcolor, C[i]);
        for(int j=graph[i];j<graph[i+1];j++)
        {
            if(C[adjList[j]] == C[i])
            {
                return 0;
            }
        }
    }
    return maxcolor;
}

void parseInput(char* inputFile, int &n, int &m, int* &graph, int* &adjList, int &D)
{
    // TODO aditya: malloc the graph here after finding the number of edges and length of array needed
    // Determine the number of colors D needed
    fstream f;
    f.open(inputFile, ios::in);
    f >> n >> m;

    graph = new int[n + 2];
    adjList = new int[2*m];
    graph[0] = m;
    vector<int> g[n + 1];
    int u,v;
    for(int i = 0; i < m; i ++)
    {
        f >> u >> v;
        g[u].push_back(v);
        g[v].push_back(u);
    }
    D = 0;
    int ctr = 0;
    for(int i = 1; i <=n; i ++)
    {
        graph[i] = ctr;
        for(auto &x: g[i])
        {
            adjList[ctr++] = x;
        }
        D = max(D, (int)g[i].size());
    }
    D++;
    graph[n + 1] = ctr;
    f.close();
    // for(int i = 0; i <2*m ; i++)
    // {
    //     cout << adjList[i] <<" ";
    // }
    // cout << endl;
    // for(int i = 0; i < n+2; i ++)
    // {
    //     cout << graph[i] <<" ";
    // }
    // cout << endl;
    return;
}

__global__ void setup_kernel(hiprandState *state){

  int idx = threadIdx.x+blockDim.x*blockIdx.x + 1;
  hiprand_init(clock64(), idx, 0, &state[idx-1]);
}

// __global__ void avgDegree1(int n, int sz, int* degree, long* ordering, int* aux_degree, int* aux_active)
// {
//     //printf("Entered avg degree\n");
//     int vert = threadIdx.x + 1;
//     if(vert > n or vert < 1)
//         return;
//     int shift = blockDim.x;
//     int su = 0;
//     int su2 = 0;
//     for(int i = vert; i <= sz;i += shift)
//     {
//         su += degree[i];
//         su2 += (ordering[i] == 0)?1:0;
//     }
//     aux_degree[vert] = su;
//     aux_active[vert] = su2;
//     //printf("The sums are %d and %d\n", su, su2);
//     // if(su2 == 0)
//     //     *avg = 0;
//     // else
//     return;
// }

// __global__ void avgDegree2(int n, int sz, int* degree, int* ordering, double* avg)
// {
//     //printf("Entered avg degree\n");
//     int vert = threadIdx.x + 1;
//     if(vert > n or vert < 1)
//         return;
//     int shift = blockDim.x;
//     int su = 0;
//     long su2 = 0;
//     for(int i = vert; i <= sz;i += shift)
//     {
//         su += degree[i];
//         su2 += ordering[i];
//     }
//     //printf("The sums are %d and %d\n", su, su2);
//     // if(su2 == 0)
//     //     *avg = 0;
//     // else
//     *avg = (double)su/su2;
//     return;
// }

__global__ void halfSum(int limit, long* arr1, long* arr2)
{
    int i = blockDim.x * blockIdx.x + threadIdx.x + 1;
    if(2*i <= limit)
    {
        arr2[i] = arr1[2*i-1]+ arr1[2*i];
    }
    else if(2*i == limit+1)
    {
        arr2[i] = arr1[2*i-1];
    }
    return;
}

__global__ void halfSum2(int limit, int* arr1, long* arr2)
{
    int i = blockDim.x * blockIdx.x + threadIdx.x + 1;
    if(2*i <= limit)
    {
        arr2[i] = arr1[2*i-1]+ arr1[2*i];
    }
    else if(2*i == limit+1)
    {
        arr2[i] = arr1[2*i-1];
    }
    return;
}

long getDegSum(int n, int* degree)
{
    long* arr1;
    if(hipMalloc(&arr1,sizeof(long)*((n+1)/2 + 1))!=hipSuccess) //(n+1)/2 for odd case handling, +1 for 1-index
    {
        cout << "Could not allocate temp_d_degree" << endl;
    }

    int threadCount = (n+1)/2;
    dim3 gridDim((threadCount+1023)/1024, 1,1);
    dim3 blockDim(1024,1,1);
    halfSum2<<<gridDim, blockDim>>>(n,degree,arr1);
    n=(n+1)/2;
    long* arr2;
    if(hipMalloc(&arr2,sizeof(long)*((n+1)/2 + 1))!=hipSuccess) //(n+1)/2 for odd case handling, +1 for 1-index
    {
        cout << "Could not allocate temp_d_degree" << endl;
    }
    long* temp;
    while(n>1)
    {
        hipDeviceSynchronize();
        threadCount = (n+1)/2;
        gridDim.x = (threadCount+1023)/1024;
        halfSum<<<gridDim, blockDim>>>(n,arr1,arr2);
        n=(n+1)/2;
        temp = arr1;
        arr1 = arr2;
        arr2 = temp;
    }
    long sum;
    auto code = hipMemcpy(&sum,&arr1[1],sizeof(long),hipMemcpyDeviceToHost);
    if (code != hipSuccess)
    {
        cout << "GPU: arr1 to sum " << hipGetErrorName(code) << " " <<  hipGetErrorString(code) << " " << endl;
    }
    // cout << "Sum : " << sum << endl;
    hipFree(arr1);
    hipFree(arr2);
    return sum;
}

__global__ void getADG(int n, double eps, double* avg, long* ordering, int* degree, hiprandState *state, int num_partition, int* temp_degree, int* graph, int* adjList)
{
    int u = blockDim.x * blockIdx.x + threadIdx.x + 1; // vertex id
    double avg_val = *avg;
    if(u > n or u < 1 or ordering[u] != 0)
    {
        return;
    }
    if(((double)degree[u]) <= avg_val*(1 + eps)) //check if need to be in current set
    {
        //Need to include this vertex in the set
        double randf = hiprand_uniform(&state[u-1]);
        double temp = scale_1 * num_partition + randf *(scale_2 + 0.99999);
        ordering[u] = (long) trunc(temp);
        // TODO BUG POSSIBLE
        atomicAdd((int*)ordering,1);
        //ordering[u] = num_partition;
        for(int i = graph[u];i<graph[u+1];i++)
        {
            atomicAdd(temp_degree+adjList[i],-1);
        }
    }
    return;
}

__global__  void updateDegree(long* ordering, int* degree, int* temp_degree, int n)
{
    int u = blockDim.x * blockIdx.x + threadIdx.x + 1; // vertex id
    if(u > n || u < 1)
    {
        return;
    }
    //already ordered
    degree[u] = (ordering[u] == 0 ? temp_degree[u] : 0);
    //Update degrees
    // for(int i = graph[u]; i < graph[u+1]; i ++)
    // {
    //     if(ordering[adjList[i]] == 0)
    //     {
    //         degree[u] += 1;
    //     }
    // }
    return;
}

__device__ int getColor(int* graph, int* adjList, long* rho, int* C, int v, int D)
{
    int deg = graph[v+1]-graph[v];
    bool* B = new bool[deg+2]();
    memset(B,0,sizeof(bool)*(deg+2));
    // if(v==n) TODO make sure n+1 th entry should be the end index of the array to make sure this works.
    // This is very important as we do not want if statement here as this function gets used a lot of times
    for(int i = graph[v]; i < graph[v+1]; i++)
    {
        if(rho[adjList[i]] > rho[v])
        {
            if(C[adjList[i]] == 0)
            {
                free(B);
                return 0;
            }
            else
            {
                if(C[adjList[i]]<=deg+1)
                {
                    B[C[adjList[i]]] = true;
                }
            }
        }
    }
    // printf("get color %d ended\n",v);
    for(int i =1 ;i <= deg+1; i++)
    {
        if(!B[i])
        {
            // printf("get color %d for %d\n",i,v);
            free(B);
            return i;
        }
    }
    // Should not come here at all
    free(B);
    assert(false);
    return 0;
}

__global__ void jpadg(int* graph, int* adjList, long* rho, int* C, int D, int n)
{
    int u = (blockDim.x * blockIdx.x)+threadIdx.x+1;
    if(u> n || u<1 || C[u]!=0)
    {
        return;
    }
    C[u] = getColor(graph, adjList, rho, C, u, D);
    if(C[u] != 0)
    {
        atomicAdd(C,1);
    }
    return;
}

long* getRho(int* graph, int* adjList, int strategy, int n)
{
    // this has to give the total order permutation on the vertices.
    // based on strategy should give random order or adg order or dec-adg order
    // Should we implement this in Device using parallel programming
    // Maybe but later

    long *rho = new long[n + 1];
    rho[0] = -1;
    for(int i  = 1; i <= n; i ++)
    {
        rho[i] = i;
    }   
    random_shuffle(rho + 1, rho + n + 1);
    return rho;
}

__global__ void getDegree(int* graph, int* degree, int n)
{
    int u = blockIdx.x * blockDim.x + threadIdx.x + 1;
    if(u>n || u<1)
    {
        return;
    }
    degree[u] = graph[u+1]-graph[u];
    return;
}

long* getRhoAdg(int* d_graph, int* d_adjList, int strategy, int n, double eps, hiprandState *d_state)
{
    int *d_degree;
    long* d_ordering;
    dim3 gridDim((n+1023)/1024,1,1);
    dim3 blockDim(1024,1,1);

    if(hipMalloc(&d_degree,sizeof(int)*(n+1))!=hipSuccess)
    {
        cout << "Could not allocate d_degree" << endl;
    }

    if(hipMalloc(&d_ordering,sizeof(long)*(n+1))!=hipSuccess)
    {
        cout << "Could not allocate d_ordering" << endl;
    }

    if(hipMemset(d_ordering, 0, sizeof(long)*(n+1)) != hipSuccess)
    {
        cout << "Could not memset d_ordering" << endl;
    }

    int num_partition = 1;
    long *ordering = new long[n+1]();
    memset(ordering, 0 , sizeof(long)*(n + 1));

    double *d_avg;
    auto code = hipMalloc(&d_avg, sizeof(double));
    if (code != hipSuccess)
    {
        cout << "GPU: Could not malloc d_avg" << hipGetErrorName(code) << " " <<  hipGetErrorString(code) << " " << endl;
    }
    code = hipMemset(d_avg, 0, sizeof(double));
    if (code != hipSuccess)
    {
        cout << "GPU:" << hipGetErrorName(code) << " " <<  hipGetErrorString(code) << " " << endl;
    }

    int* temp_d_degree;
    if(hipMalloc(&temp_d_degree,sizeof(int)*(n+1))!=hipSuccess)
    {
        cout << "Could not allocate temp_d_degree" << endl;
    }

    getDegree<<<gridDim, blockDim>>>(d_graph, temp_d_degree, n);
    ordering[0] = 0;
    while(ordering[0]<n)
    {
        // cout <<"Finished ordering" << ordering[0] << endl;
        double *avg = new double;

        updateDegree<<<gridDim,blockDim>>>(d_ordering, d_degree, temp_d_degree, n);
        hipDeviceSynchronize();

        *avg = double(getDegSum(n,d_degree))/(n-ordering[0]);

        code = hipMemcpy(d_avg, avg, sizeof(double),hipMemcpyHostToDevice);
        if (code != hipSuccess)
        {
            cout << "GPU: d_avg to avg " << hipGetErrorName(code) << " " <<  hipGetErrorString(code) << " " << endl;
        }
        // cout <<"avg degree is "<<*avg << endl;
        getADG<<<gridDim, blockDim>>>(n, eps, d_avg, d_ordering, d_degree, d_state, num_partition++, temp_d_degree, d_graph, d_adjList);
        code = hipMemcpy(ordering,d_ordering,sizeof(long)*(n+1),hipMemcpyDeviceToHost); // copy from device to host
        if (code != hipSuccess)
        {
            cout << "GPU d_ordering into ordering " << hipGetErrorName(code) << " " <<  hipGetErrorString(code) << " " << endl;
        }
    }
    assert(ordering[0]==n);
    hipFree(d_degree);
    hipFree(temp_d_degree);
    hipFree(d_avg);
    hipFree(d_ordering);
    // cout <<"Finished ordering " << ordering[0] << endl;
    return ordering;
}


int main(int argc, char** argv)
{
    if(argc == 1)
    {
        cout << "No input" << endl;
        return 0;
    }
    int n, m, D;
    int *adjList = NULL;
    int *graph = NULL;
    parseInput(argv[1], n , m, graph, adjList, D);
    // cout << "Parse input D " << D << " n " << n << " m " << m << endl;
    int *d_graph, *d_adjList;
    
    if(hipMalloc(&d_graph,sizeof(int)*(n+2))!=hipSuccess)
    {
        cout << "Could not allocate d_graph" << endl;
    }

    if(hipMalloc(&d_adjList, sizeof(int)*(2*m))!=hipSuccess)
    {
        cout << "Could not allocate d_adjList" << endl;
    }

    if(hipMemcpy(d_graph,graph,sizeof(int)*(n+2),hipMemcpyHostToDevice) != hipSuccess){
        cout<<"Could not copy graph into d_graph"<<endl;
    }

    if(hipMemcpy(d_adjList,adjList,sizeof(int)*(2*m),hipMemcpyHostToDevice) != hipSuccess){
        cout<<"Could not copy adjList into d_adjList"<<endl;
    }

    dim3 gridDim((n+1023)/1024,1,1);
    dim3 blockDim(1024,1,1);
    hiprandState *d_state;
    hipMalloc(&d_state, blockDim.x * gridDim.x * sizeof(hiprandState));
    setup_kernel<<<gridDim,blockDim>>>(d_state);
    // cout <<"finished random number generation" << endl;

    //Start cuda timer after init
    hipEvent_t start, stop;
    hipEventCreate(&start);
    hipEventCreate(&stop);
    hipEventRecord(start);

    const double eps = 0.5;

    // cout <<"calling getrhoadg" << endl;
    long *rho = getRhoAdg(d_graph, d_adjList, 0, n, eps, d_state);

    //print ADG ordering
    map<long,long> mymap;
    long flag = 0;
    for(int i = 1; i <= n; i ++)
    {
        while(mymap.find(rho[i]) != mymap.end())
        {
            rho[i]++;
            flag++;
        }
        mymap[rho[i]]++;
    }
    cout <<"Number of collisions: " << flag  << "/" << n << endl;

    long* d_rho;
    if(hipMalloc(&d_rho,sizeof(long)*(n+1))!=hipSuccess)
    {
        cout << "Could not allocate d_graph" << endl;
    }

    if(hipMemcpy(d_rho,rho,sizeof(long)*(n+1),hipMemcpyHostToDevice) != hipSuccess){
        cout<<"Could not copy rho into d_rho"<<endl;
    }
    free(rho);

    int* C = new int[n+1](); //change back to int if needed
    memset(C, 0, sizeof(int)*(n+1));
    int* d_C;
    if(hipMalloc(&d_C, sizeof(int)*(n+1))!=hipSuccess)
    {
        cout << "Could not allocate d_C" << endl;
    }
    if(hipMemset(d_C, 0, sizeof(int)*(n+1)) != hipSuccess)
    {
        cout << "Could not memset C" << endl;
    }

    // C[0] = 0;
    int iter = 0;
    while(C[0]< n)
    {
        // We need not run again for all vertices
        // Run only for uncolored vertices VERY IMPORTANT
        // cout << "Running iteration " << iter++ << " colored : " << C[0] << "/" << n << endl;
        iter++;
        jpadg<<<gridDim, blockDim>>>(d_graph, d_adjList, d_rho, d_C, D, n);
        auto code = hipMemcpy(C,d_C,sizeof(int)*(n+1),hipMemcpyDeviceToHost);
        if (code != hipSuccess)
        {
            cout << "GPU:" << hipGetErrorName(code) << " " <<  hipGetErrorString(code) << " " << endl;
        }
    }
    // cout << "Running iteration " << iter++ << " colored : " << C[0] << "/" << n << endl;

    hipEventRecord(stop);
    hipEventSynchronize(stop);
    float milliseconds = 0;
    hipEventElapsedTime(&milliseconds, start, stop);

    int maxcolor = checkValidColoring(graph, adjList, C, n);
    if(maxcolor == 0)
    {
        cout << "coloring wrong" << endl;
    }
    cout << "n: " << n << " m: " << m << " D: " << D << " maxColor: " << maxcolor << " time(ms): " << milliseconds << " iter: " << iter << endl;
    hipEventDestroy(start);
    hipEventDestroy(stop);

    hipFree(d_graph);
    hipFree(d_adjList);
    hipFree(d_rho);
    hipFree(d_C);

    free(graph);
    free(adjList);
    free(C);
    return 0;
}
