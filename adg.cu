#include "hip/hip_runtime.h"
#include <iostream>
#include <math.h>
#include <algorithm>
#include <cassert>
#include <string>
#include<ifstream>

using namespace std;
using std::min;
using std::max;




void parseInput(string inputFile, int &n, int &m, int *adjList, int* graph, int &D)
{
    // TODO aditya: malloc the graph here after finding the number of edges and length of array needed
    // Determine the number of colors D needed
    ifstream f;
    f.open(inputFile.c_str(), ios::in);
    f >> n >> m;

    graph = new int[n + 2];
    adjList = new int[2*m];
    graph[0] = m;
    vector<int> g[n + 1];
    int u,v;
    for(int i = 0; i < m; i ++)
    {
        f >> u >> v;
        g[u].push_back(v);
        g[v].push_back(u);
    }
    D = 0;
    int ctr = 0;
    for(int i = 1; i <=n; i ++)
    {
        graph[i] = ctr;
        for(auto &x: g[i])
        {
            adjList[ctr++] = x;
        }
        D = max(D, (int)g[i].size());
    }
    graph[n + 1] = ctr;
    for(int i = 0; i <=n ; i++)
    {
        cout << adjList[i] <<" ";
    }
    cout << endl;
    for(int i = 0; i < m; i ++)
    {
        cout << graph[i] <<" ";
    }
    cout << endl;

}
int* getrho(int* graph, int* adjList, int strategy, int n)
{
    // this has to give the total order permutation on the vertices.
    // based on strategy should give random order or adg order or dec-adg order
    // Should we implement this in Device using parallel programming
    // Maybe but later

    int *rho = new int[n + 1];
    rho[0] = -1;
    for(int i  = 1; i <= n; i ++)
    {
        rho[i] = i;
    }
    random_shuffle(rho + 1, rho + n + 1);
    return rho;

int main()
{
    if(argc == 1)
    {
        cout << "No input" << endl;
    }
    // TODO by aditya
    int n, m, D;
    int *adjList; //This is the adjacency list
    int *graph;
    parseInput(argv[1], n , m, adjList, graph, D);

    int* rho = getRho(graph, 1); // 1= random order or largest degree first
    
    dim3 gridDim((n+1023)/1024,1,1);
    dim3 blockDim(1024,1,1);

    int *d_graph, *d_adjList, *d_rho, *d_C;

    if(hipMalloc(&d_graph,sizeof(int)*(n+2))!=hipSuccess)
    {
        cout << "Could not allocate d_graph" << endl;
    }

    if(hipMalloc(&d_adjList, sizeof(int)*(2*m))!=hipSuccess)
    {
        cout << "Could not allocate d_adjList" << endl;
    }

    if(hipMalloc(&d_rho,sizeof(int)*n)!=hipSuccess)
    {
        cout << "Could not allocate d_graph" << endl;
    }

    if(hipMalloc(&d_C, sizeof(int)*n)!=hipSuccess)
    {
        cout << "Could not allocate d_C" << endl;
    }

    if(hipMemcpy(d_graph,graph,sizeof(int)*(n+2),hipMemcpyHostToDevice) != hipSuccess){
        cout<<"Could not copy graph into d_graph"<<endl;
    }

    if(hipMemcpy(d_adjList,adjList,sizeof(int)*(2*m),hipMemcpyHostToDevice) != hipSuccess){
        cout<<"Could not copy adjList into d_adjList"<<endl;
    }

    if(hipMemcpy(d_rho,rho,sizeof(int)*(n),hipMemcpyHostToDevice) != hipSuccess){
        cout<<"Could not copy rho into d_rho"<<endl;
    }

    if(hipMemset(&d_C, 0, sizeof(int)*n) != hipSuccess)
    {
        cout << "Could not memset C" << endl;
    }

    while(notAllVerticesColored(C,n))
    {
        // We need not run again for all vertices
        // Run only for uncolored vertices VERY IMPORTANT
        jpadg<<<gridDim, blockDim>>>(d_graph, d_adjList, d_rho, d_C);
        if(hipMemcpy(C,d_C,sizeof(int)*n,hipMemcpyDeviceToHost) != hipSuccess)
        {
            cout << "Could not copy d_C into C" << endl;
        }
    }

    assert(true);
    // assert(checkValidColoring(graph, adjList, C));
    return 0;
}

__device__ int getColor(int* graph, int* rho, int* C, int v, int D)
{
    bool B[D] = 0;
    // if(v==n) TODO make sure n+1 th entry should be the end index of the array to make sure this works.
    // This is very important as we do not want if statement here as this function gets used a lot of times
    for(int i = graph[v]; i < graph[v+1]; i++)
    {
        if(rho[adjList[i]] > rho[v])
        {
            if(C[adjList[i]] == -1)
            {
                return -1;
            }
            else
            {
                B[C[adjList[i]]] = true;
            }
        }
    }
    for(int i =1 ;i <= D; i++)
    {
        if(!B[i])
        {
            return i;
        }
    }
    // Should not come here at all
    assert(false);
    return -1;
}

__global__ void jpadg(int* graph, int* rho, int* C)
{
    int u = (blockDim.x * blockIdx.x)+threadIdx.x;
    int minC = -1;
    // A very important change to make while loop to if. We just give one chance if doesn't get color
    // give another chance again later, no point in giving a chance again immediately
    if(C[u] == -1)
    {
        minC = getColor(u);
        if(minC != -1)
        {
            C[u] = minC;
        }
    }
    return;
}
