#include "hip/hip_runtime.h"
#include <iostream>
#include <math.h>
#include <algorithm>
#include <cassert>
#include <string>

using namespace std;
using std::min;
using std::max;

int* parseInput(string inputFile)
{
    // TODO aditya: malloc the graph here after finding the number of edges and length of array needed
    // Determine the number of colors D needed
}

int main()
{
    if(argc == 1)
    {
        cout << "No input" << endl;
    }
    // TODO by aditya
    int* graph = parseInput(argv[1]);
    
}

__device__ int getColor(int* graph, int* rho, int* C, int v, int D)
{
    bool B[D] = 0;
    // if(v==n) TODO make sure n+1 th entry should be the end index of the array to make sure this works.
    // This is very important as we do not want if statement here as this function gets used a lot of times
    for(int i = graph[v];i<graph[v+1];i++)
    {
        if(rho[graph[i]] > rho[v])
        {
            if(C[graph[i]] == -1)
            {
                return -1;
            }
            else
            {
                B[C[graph[i]]] = true;
            }
        }
    }
    for(int i =0;i<D;i++)
    {
        if(!B[i])
        {
            return i;
        }
    }
    // Should not come here at all
    assert(false);
    return -1;
}

__global__ void jpadg(int* graph, int* rho, int* C)
{
    int u = (blockDim.x * blockIdx.x)+threadIdx.x;
    int minC = -1;
    // A very important change to make while loop to if. We just give one chance if doesn't get color
    // give another chance again later, no point in giving a chance again immediately
    if(C[u] == -1)
    {
        minC = getColor(u);
        if(minC != -1)
        {
            C[u] = minC;
        }
    }
    return;
}