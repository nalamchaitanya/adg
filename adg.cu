#include "hip/hip_runtime.h"
#include <iostream>
#include <math.h>
#include <algorithm>
#include <cassert>
#include <string>
#include<ifstream>

using namespace std;
using std::min;
using std::max;




void parseInput(string inputFile, int &n, int &m, int *adjList, int* graph, int &D)
{
    // TODO aditya: malloc the graph here after finding the number of edges and length of array needed
    // Determine the number of colors D needed
    ifstream f;
    f.open(inputFile.c_str(), ios::in);
    f >> n >> m;

    graph = new int[n + 2];
    adjList = new int[2*m];
    graph[0] = m;
    vector<int> g[n + 1];
    int u,v;
    for(int i = 0; i < m; i ++)
    {
        f >> u >> v;
        g[u].push_back(v);
        g[v].push_back(u);
    }
    D = 0;
    int ctr = 0;
    for(int i = 1; i <=n; i ++)
    {
        graph[i] = ctr;
        for(auto &x: g[i])
        {
            adjList[ctr++] = x;
        }
        D = max(D, (int)g[i].size());
    }
    graph[n + 1] = ctr;
    for(int i = 0; i <=n ; i++)
    {
        cout << adjList[i] <<" ";
    }
    cout << endl;
    for(int i = 0; i < m; i ++)
    {
        cout << graph[i] <<" ";
    }
    cout << endl;

}
int* getrho(int* graph, int* adjList, int strategy, int n)
{
    // this has to give the total order permutation on the vertices.
    // based on strategy should give random order or adg order or dec-adg order
    // Should we implement this in Device using parallel programming
    // Maybe but later

    int *rho = new int[n + 1];
    rho[0] = -1;
    for(int i  = 1; i <= n; i ++)
    {
        rho[i] = i;
    }
    random_shuffle(rho + 1, rho + n + 1);
    return rho;

int main()
{
    if(argc == 1)
    {
        cout << "No input" << endl;
    }
    // TODO by aditya
    int n, m, D;
    int *adjList; //This is the adjacency list
    int *graph;
    parseInput(argv[1], n , m, adjList, graph, D);

    int* rho = getRho(graph, 1); // 1= random order or largest degree first
    
    dim3 gridDim((n+1023)/1024,1,1);
    dim3 blockDim(1024,1,1);

    int *d_graph, *d_rho, *d_C;

    if(hipMalloc(&d_graph,))
    
}

__device__ int getColor(int *graph, int* adjList, int* rho, int* C, int v, int D) //a is the adjacency list mapping
{
    bool *B = new bool[D + 1] ();
    // if(v==n) TODO make sure n+1 th entry should be the end index of the array to make sure this works.
    // This is very important as we do not want if statement here as this function gets used a lot of times
    for(int i = graph[v]; i < graph[v+1]; i++)
    {
        if(rho[adjList[i]] > rho[v])
        {
            if(C[adjList[i]] == -1)
            {
                return -1;
            }
            else
            {
                B[C[adjList[i]]] = true;
            }
        }
    }
    for(int i =1 ;i <= D; i++)
    {
        if(!B[i])
        {
            return i;
        }
    }
    // Should not come here at all
    assert(false);
    return -1;
}

__global__ void jpadg(int* graph, int* rho, int* C)
{
    int u = (blockDim.x * blockIdx.x)+threadIdx.x;
    int minC = -1;
    // A very important change to make while loop to if. We just give one chance if doesn't get color
    // give another chance again later, no point in giving a chance again immediately
    if(C[u] == -1)
    {
        minC = getColor(u);
        if(minC != -1)
        {
            C[u] = minC;
        }
    }
    return;
}
