
#include <hip/hip_runtime.h>
#include <iostream>
#include <math.h>
#include <algorithm>
#include <cassert>

using namespace std;
using std::min;
using std::max;

__device__ int IJtoIdx(int i, int j)
{
    int t_blockIdx_x = i/blockDim.x;
    int t_threadIdx_x = i % blockDim.x;
    int t_blockIdx_y = j/blockDim.y;
    int t_threadIdx_y = j % blockDim.y;
    int topCorner = (t_blockIdx_x * gridDim.y + t_blockIdx_y)*blockDim.x*blockDim.y;
    int idx = topCorner + t_threadIdx_x*blockDim.y + t_threadIdx_y;
    return idx;
}

int Host_IJtoIdx(int i, int j, int b_x, int b_y, int g_x, int g_y)
{
    int t_blockIdx_x = i/b_x;
    int t_threadIdx_x = i % b_x;
    int t_blockIdx_y = j/b_y;
    int t_threadIdx_y = j % b_y;
    int topCorner = (t_blockIdx_x * g_y + t_blockIdx_y)*b_x*b_y;
    int idx = topCorner + t_threadIdx_x*b_y + t_threadIdx_y;
    return idx;
}

__device__ double secondmin(double a, double b, double c, double d)
{
    return max(min(min(a,b), max(c,d)), min(max(a,b), min(c,d)));
}

__global__ void Stencil(double *a, double *b, int n)
{
    int i = blockIdx.x*blockDim.x + threadIdx.x;
    int j = blockIdx.y*blockDim.y + threadIdx.y;
    int idx = IJtoIdx(i,j);
    if(i == 0 || i >= n-1 || j == 0 || j >= n-1)
    {
        b[idx] = a[idx];
    }
    else
    {
        // Can be optimized later by just writing all this in one function to find second min
        double top = a[IJtoIdx(i-1,j-1)];
        double bottom = a[IJtoIdx(i+1,j+1)];
        double left = a[IJtoIdx(i+1,j-1)];
        double right = a[IJtoIdx(i-1,j+1)];
        b[idx] = a[idx] + secondmin(top,bottom,left,right);
    }
    return;
}

__global__ void sumElements1(double *arr, int sz)
{
    int st = threadIdx.x;
    if(st >= sz)
        return;
    int shift = blockDim.x;
    double su = 0;
    for(int i = st; i < sz;i += shift)
    {
        su += arr[i];
    }
    arr[st] = su;
}

__global__ void sumElements2(double *arr, int num)
{
    if(num == 0)
        return;
    double su = 0;
    for(int i = 0; i < num; i++)
    {
        su += arr[i];
    }
    arr[0] = su;
}

void init(double *A, int n, int g_x, int g_y, int b_x, int b_y)
{
    double temp = 0;
    for(int i = 0;i< n;i++)
    {
        for(int j = 0;j<n;j++)
        {
            temp = 1+cos(2*i)+sin(j);
            A[Host_IJtoIdx(i,j,b_x,b_y,g_x,g_y)] = temp*temp;
        }
    }
    return;
}

int main(int argc, char **argv)
{
    //allocate memory for device variables
    double *d_A,*d_B;

    int A_x = atoi(argv[1]);
    int A_y = atoi(argv[1]);
    
    int b_x = 32;
    int b_y = 32;

    int g_x = (A_x + b_x - 1)/b_x;
    int g_y = (A_y + b_y - 1)/b_y;

    dim3 gridDim(g_x,g_y,1);
    dim3 blockDim(b_x,b_y,1);

    int A_size = g_x*g_y*b_x*b_y;

    // Allocating memory on host
    double *A = new double[A_size];
    memset(A, 0, sizeof(double)*A_size);

    //we can check if the cuda functions fail by seeing if they return a cudaSuccess code
    //you get status codes like cudaSuccess for free when you are compiling with nvcc
    if(hipMalloc(&d_A,sizeof(double)*A_size) != hipSuccess){
        cout<<"Could not allocate d_A"<<endl;
    }
    hipMemset(&d_A ,0,A_size*sizeof(double));
    if(hipMalloc(&d_B,sizeof(double)*A_size) != hipSuccess){
        cout<<"Could not allocate d_A"<<endl;
    }
    hipMemset(&d_B ,0,A_size*sizeof(double));

    // Init<<<gridDim,blockDim>>>(d_A, b_x*b_y, A_x);
    init(A,A_x,g_x,g_y, b_x,b_y);
    if(hipMemcpy(d_A,A,sizeof(double)*A_size,hipMemcpyHostToDevice) != hipSuccess){
        cout<<"Could not copy A into d_A"<<endl;
    }

    //Start cuda timer after init
    hipEvent_t start, stop;
    hipEventCreate(&start);
    hipEventCreate(&stop);
    hipEventRecord(start);



    int t = 10;
    for(int iter = 0;iter < t;iter++)
    {
        hipDeviceSynchronize();
        if(iter%2 == 0)
        {
            Stencil<<<gridDim, blockDim>>>(d_A, d_B, A_x);
        }
        else
        {
            Stencil<<<gridDim, blockDim>>>(d_B, d_A, A_x);
        }
    }

    double sum, val;
    int val_idx;
    if(A_x >= 48)
    {
        val_idx = Host_IJtoIdx(37,47,b_x,b_y,g_x,g_y);
        if(hipMemcpy(&val,d_A+val_idx,sizeof(double),hipMemcpyDeviceToHost) != hipSuccess)
        {
            cout<<"Could not copy d_A into A"<<endl;
        }
    }
    else
    {
        val_idx = -1;
    }

    hipDeviceSynchronize();
    sumElements1<<<1,1024>>>(d_A, A_size);

    hipDeviceSynchronize();
    sumElements1<<<1,1>>>(d_A, min(1024, A_size));

    hipEventRecord(stop);
    hipEventSynchronize(stop);
    float milliseconds = 0;
    hipEventElapsedTime(&milliseconds, start, stop);

    if(hipMemcpy(&sum,d_A,sizeof(double),hipMemcpyDeviceToHost) != hipSuccess){
        cout<<"Could not copy d_A into A"<<endl;
    }

    hipFree(d_A);
    hipFree(d_B);
    hipEventDestroy(start);
    hipEventDestroy(stop);
    if(val_idx == -1)
    {
        cout << "n: " << A_x << " sum: " << sum << " val: N/A time(ms): " << milliseconds << endl;
    }
    else
    {
        cout << "n: " << A_x << " sum: " << sum << " val: " << val << " time(ms): " << milliseconds << endl;
    }

    return 0;
}